#include <iostream>
#include <hip/hip_runtime.h>

#define M 2000
#define N 1000

#define CHECK(call)\
{\
    const hipError_t error=call;\
    if(error!=hipSuccess)\
    {\
        printf("ERROR: %s:%d,",__FILE__,__LINE__);\
        printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
        exit(1);\
    }\
}

using namespace std;

__global__ void m_add(float *c, float *a, float *b, int m, int n){
    // get ind
    const int block_ind = blockIdx.x;
    const int thread_ind = threadIdx.x;
    const int step = blockDim.x * gridDim.x;
    int curr_ind = block_ind * blockDim.x + thread_ind;

    while (curr_ind < m * n) {
        c[curr_ind] = a[curr_ind] + b[curr_ind];
	curr_ind += step;
    }
}

int main() {
    float *h_a, *h_b, *h_o;
    float *d_a, *d_b, *d_o;

    // define timer
    hipEvent_t start, stop;

    // initialize data
    h_a = (float*) malloc(sizeof(float) * (M * N));
    h_b = (float*) malloc(sizeof(float) * (M * N));
    h_o = (float*) malloc(sizeof(float) * (M * N));

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            int ind = i * N + j;
	    h_a[ind] = ind + 324.4553f;
	    h_b[ind] = ind + 2.3232f;
	}
    }

    // GPU memory allco
    CHECK(hipMalloc((void**) &d_a, sizeof(float) * (M * N)));
    CHECK(hipMalloc((void**) &d_b, sizeof(float) * (M * N)));
    CHECK(hipMalloc((void**) &d_o, sizeof(float) * (M * N)));

    // copy from CPU to GPU
    CHECK(hipMemcpy(d_a, h_a, sizeof(float) * (M * N), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(float) * (M * N), hipMemcpyHostToDevice));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // cuda add
    m_add<<<M, N>>>(d_o, d_a, d_b, M, N);
    CHECK(hipDeviceSynchronize());
    // copy results back to CPU
    CHECK(hipMemcpy(h_o, d_o, sizeof(float) * (M * N), hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "The kernel function is during: " << elapsedTime << endl;

    cout << "The final number (M * N): " << h_o[234];
    // free GPU mem
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_o);
    free(h_a);
    free(h_b);
    hipDeviceReset();
    return 0;

}
