
#include <hip/hip_runtime.h>
#include <iostream>

#define M 2000
#define N 1000

using namespace std;

__global__ void m_add(float *c, float *a, float *b, int m, int n){
    // get ind
    const int block_ind = blockIdx.x;
    const int thread_ind = threadIdx.x;
    const int step = blockDim.x * gridDim.x;
    int curr_ind = block_ind * blockDim.x + thread_ind;

    while (curr_ind < m * n) {
        c[curr_ind] = a[curr_ind] + b[curr_ind];
	curr_ind += step;
    }
}

int main() {
    float *h_a, *h_b, *h_o;
    float *d_a, *d_b, *d_o;

    // define timer
    hipEvent_t start, stop;

    // initialize data
    h_a = (float*) malloc(sizeof(float) * (M * N));
    h_b = (float*) malloc(sizeof(float) * (M * N));
    h_o = (float*) malloc(sizeof(float) * (M * N));

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            int ind = i * N + j;
	    h_a[ind] = ind + 324.4553f;
	    h_b[ind] = ind + 2.3232f;
	}
    }

    // GPU memory allco
    hipMalloc((void**) &d_a, sizeof(float) * (M * N));
    hipMalloc((void**) &d_b, sizeof(float) * (M * N));
    hipMalloc((void**) &d_o, sizeof(float) * (M * N));

    // copy from CPU to GPU
    hipMemcpy(d_a, h_a, sizeof(float) * (M * N), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * (M * N), hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // cuda add
    m_add<<<1, 1>>>(d_o, d_a, d_b, M, N);

    // copy results back to CPU
    hipMemcpy(h_o, d_o, sizeof(float) * (M * N), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "The kernel function is during: " << elapsedTime << endl;

    cout << "The final number (M * N): " << h_o[234];
    // free GPU mem
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_o);
    free(h_a);
    free(h_b);

    return 0;

}
